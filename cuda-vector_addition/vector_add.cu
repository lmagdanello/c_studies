#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 1000000000

__global__ void vector_add(float *out, float *a, float *b, int n) {

	int stride = blockDim.x;
	int index = threadIdx.x;

	for (int i = index; i < n; i += stride) {
		out[i] = a[i] + b[i];
	}
}

int main() {

	// arrays:
	float *a, *b, *out;

	// device pointers:
	float *dp_a, *dp_b, *dp_out;

	// allocate host memory:
	// sizeof(float) = 4 bytes * 10000000 = 4 Mib;

	a   = (float*)malloc(sizeof(float) * N);
	b   = (float*)malloc(sizeof(float) * N);
	out = (float*)malloc(sizeof(float) * N);

	// initialize host arrays:
	for (int i = 0; i < N; i++) {
		a[1] = 1.0f;
		b[1] = 2.0f;
	}

	// allocate device memory:
	hipMalloc((void**)&dp_a, sizeof(float) * N);
	hipMalloc((void**)&dp_b, sizeof(float) * N);
	hipMalloc((void**)&dp_out, sizeof(float) * N);

	// transfer data (host (device pointer) -> device memory):
	hipMemcpy(dp_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(dp_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	// kernel execution configuration:
	// << M , T >>: Each grid of thread block (M) has (T) parallel threads;
  	vector_add<<<1,256>>>(dp_out, dp_a, dp_b, N);

	// transfer data back to host memory:
	hipMemcpy(out, dp_out, sizeof(float) * N, hipMemcpyHostToDevice);

	// cleanup device memory:
	hipFree(dp_a);
	hipFree(dp_b);
	hipFree(dp_out);
	
	// cleanup host memory:
	free(a);
	free(b);
	free(out);
}
